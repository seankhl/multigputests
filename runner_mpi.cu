#include "hip/hip_runtime.h"

#include <vector>
#include <random>
#include <algorithm>
#include <functional>
#include <iostream>
#include <chrono>
//#include "Eigen/dense"

#include "sim.cuh"

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

const int X_SZ = 2048;
const int Y_SZ = 2048;
const float RANGE = 32;
const int natoms = 1000;

void checkDevices(int ngpus) {
    /* Check capability of the GPU 
       (should be done for each card to be used)
    */
    std::vector<hipDeviceProp_t> gpuprops(ngpus);
    bool is_fermi = true;
    bool has_uva = true;
    // second argument is gpu number
    for (int i = 0; i < ngpus; ++i) {
        hipGetDeviceProperties(&gpuprops[i], i + 1);
    }
    // check results
    for (int i = 0; i < ngpus; ++i) {
        is_fermi &= (gpuprops[i].major >= 2); // must be Fermi based
        has_uva &= (gpuprops[i].unifiedAddressing); // must be Fermi based
    }
    
    // TODO: only works for ngpus == 2
    int access2from1, access1from2;
    hipDeviceCanAccessPeer(&access2from1, 2, 1);
    hipDeviceCanAccessPeer(&access1from2, 1, 2);
    bool same_complex = (access2from1 && access1from2);
    std::cout << "num devices: " << ngpus << 
                 "; is fermi? " << is_fermi << 
                 "; has uva? " << has_uva <<
                 "; same complex? " << same_complex << std::endl;
}

int run(int ngpus_in, int nt)
{
    //constexpr int X_NBINS = X_SZ / RANGE + 1;
    //constexpr int Y_NBINS = Y_SZ / RANGE + 1;
    //Grid grid = new Grid(X_NBINS, Y_NBINS);

    int ngpus;
    hipGetDeviceCount(&ngpus);

    if (ngpus_in < ngpus) { ngpus = ngpus_in; }
    checkDevices(ngpus);

    // random initial values for atoms
    std::random_device rd;
    std::mt19937 mt_rand(rd());
    std::mt19937::result_type x_seed = time(0);
    auto x_rand = std::bind(
                        std::uniform_real_distribution<float>(0, X_SZ), 
                        std::mt19937(1));
    std::mt19937::result_type y_seed = time(0);
    auto y_rand = std::bind(
                        std::uniform_real_distribution<float>(0, Y_SZ), 
                        std::mt19937(2));
    std::mt19937::result_type val_seed = time(0);
    auto val_rand = std::bind(
                        std::uniform_real_distribution<float>(0, 1), 
                        std::mt19937(3));

    // create atoms and store them in the grid
    std::array<float4, natoms> atoms;
    for (int i = 0; i < natoms; ++i) {
        atoms[i].x = x_rand();
        atoms[i].y = y_rand();
        atoms[i].z = val_rand();
        //grid.record_atom(atoms[atoms.size() - 1]);
    }

    std::sort(atoms.begin(), atoms.end(), [](float4 a, float4 b) {
        return a.y < b.y;
    });

    int atoms_off = 0;
    
    std::vector<int> cutlo(ngpus, 0);
    std::vector<int> atoms_split(ngpus, natoms);
    std::vector<int> atoms_width(ngpus, natoms);
    std::vector<int> cuthi(ngpus, natoms);
    for (int i = 0; i < ngpus; ++i) {
        for (int j = 0; j < natoms; ++j) {
            if (atoms[j].y > Y_SZ/ngpus * (i+1)) {
                atoms_split[i] = j;
                atoms_width[i] = j;
                break;
            }
        }
    }
    for (int i = 1; i < ngpus; ++i) {
        atoms_width[i] -= atoms_width[i-1];
    }
    for (int i = 0; i < ngpus; ++i) {
        for (int j = 0; j < natoms; ++j) {
            if (atoms[j].y > (Y_SZ/ngpus - RANGE) * (i+1)) {
                cutlo[i] = j;
                break;
            }
        }
    }
    for (int i = 0; i < ngpus; ++i) {
        for (int j = 0; j < natoms; ++j) {
            if (atoms[j].y > (Y_SZ/ngpus + RANGE) * (i+1)) {
                cuthi[i] = j;
                break;
            }
        }
    }
    for (int i = 0; i < ngpus; ++i) {
        std::cout << "gpu id: " << i+1 << 
                     " cutlo: " << cutlo[i] <<
                     " split: " << atoms_split[i] <<
                     " cuthi: " << cuthi[i] << std::endl;
    }

    std::vector<float4 *> atoms_old_dev(ngpus);
    std::vector<float4 *> atoms_new_dev(ngpus);
    for (int i = 0; i < ngpus; ++i) {
        hipSetDevice(i + 1);
        
        // new vals: before timestepping, current vals always in here
        HANDLE_ERROR( hipMalloc((void **)&atoms_new_dev[i], 
                                 atoms_width[i] * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)atoms_new_dev[i], 
                                 (void *)(atoms.data() + atoms_off), 
                                 atoms_width[i] * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
        
        // malloc space for old vals
        HANDLE_ERROR( hipMalloc((void **)&atoms_old_dev[i], 
                                 atoms_width[i] * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)atoms_old_dev[i], 
                                 (void *)(atoms.data() + atoms_off), 
                                 atoms_width[i] * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
        
        
        atoms_off += atoms_width[i];
    }
    
    std::vector<float4 *> ghost_lo_dev(ngpus-1);
    std::vector<float4 *> ghost_hi_dev(ngpus-1);
    for (int i = 0; i < ngpus-1; ++i) {  // don't need last split, always end
        // ghost vals
        hipSetDevice(i + 2);  // next proc gets lo ghosts
        HANDLE_ERROR( hipMalloc((void **)&ghost_lo_dev[i], 
                                 (atoms_split[i] - cutlo[i]) * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)ghost_lo_dev[i], 
                                 (void *)(atoms.data() + cutlo[i]), 
                                 (atoms_split[i] - cutlo[i]) * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
        hipSetDevice(i + 1);  // this proc gets hi ghosts
        HANDLE_ERROR( hipMalloc((void **)&ghost_hi_dev[i], 
                                 (cuthi[i] - atoms_split[i]) * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)ghost_hi_dev[i], 
                                 (void *)(atoms.data() + atoms_split[i]), 
                                 (cuthi[i] - atoms_split[i]) * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
    }
    
    // timestep
    //int x_cell = 0;
    //int y_cell = 0;
    std::cout << "num timesteps: " << nt << std::endl;
    for (int i = 0; i < natoms; i += natoms/10) {
        std::cout << atoms[i].x << " " << 
                     atoms[i].y << " " << 
                     atoms[i].z << std::endl;
    }

    float4 *needs_lo;
    int needs_lo_sz;
    float4 *needs_hi;
    int needs_hi_sz;
    float4 *atoms_tmp_dev;
    for (int t = 0; t < nt; ++t) {
        for (int i = 0; i < ngpus; ++i) {
            hipSetDevice(i + 1);
            //if (t % 10000 == 0) { std::cout << t << std::endl; }

            // figure out what our needed ghosts are
            if (i == 0) {
                needs_lo = NULL;
                needs_lo_sz = 0;
            } else {
                needs_lo = ghost_lo_dev[i-1];
                needs_lo_sz = atoms_split[i-1] - cutlo[i-1];
            }
            if (i == ngpus-1) {
                needs_hi = NULL;
                needs_hi_sz = 0;
            } else {
                needs_hi = ghost_hi_dev[i];
                needs_hi_sz = cuthi[i] - atoms_split[i];
            }
            
            // swap old and new pointers
            atoms_tmp_dev = atoms_new_dev[i];
            atoms_new_dev[i] = atoms_old_dev[i];
            atoms_old_dev[i] = atoms_tmp_dev;

            // run sim
            timestep<<<atoms_width[i], 1>>>(
                        atoms_width[i], RANGE, 
                        atoms_old_dev[i], atoms_new_dev[i],
                        needs_lo, needs_lo_sz,
                        needs_hi, needs_hi_sz);

            hipDeviceSynchronize();

            // update ghosts
            if (i != 0) {
                int ghost_lo_sz = atoms_split[i-1] - cutlo[i-1];
                if (ghost_lo_sz != 0) {  // i != ngpus-1
                    HANDLE_ERROR( 
                        hipMemcpy(
                            (void *)ghost_lo_dev[i-1], 
                            (void *)(atoms_new_dev[i-1] + 
                                        (atoms_width[i-1] - ghost_lo_sz)), 
                            ghost_lo_sz * sizeof(float4), 
                            hipMemcpyDeviceToDevice) );
                }
                int ghost_hi_sz = cuthi[i-1] - atoms_split[i-1];
                if (cuthi[i-1] - atoms_split[i-1] != 0) {  // i != 0
                    HANDLE_ERROR( 
                        hipMemcpy(
                            (void *)ghost_hi_dev[i-1], 
                            (void *)(atoms_new_dev[i]), 
                            ghost_hi_sz * sizeof(float4), 
                            hipMemcpyDeviceToDevice) );
                }
            }

        }
        /*
        for (auto atom: atoms) {
            grid.get_cell_by_atom(atom, x_cell, y_cell);
            for (int x_off = -1; x_off <= 1; ++x_off) {
                for (int y_off = -1; y_off <= 1; ++y_off) {
                    vector<int> neighbors = grid.get_atoms_in_cell(
                                                x_cell + x_off, 
                                                y_cell + y_off);
                    if (x_off != 0 && y_off != 0) {
                        neighbors
        */
    }
    /*
            timestep<<<atoms_width[i], 1>>>(
                        atoms_width[i], RANGE, 
                        atoms_new_dev[i], atoms_old_dev[i],
                        ghost_lo_dev[i], atoms_split[i] - cutlo[i],
                        ghost_hi_dev[i], cuthi[i] - atoms_split[i]);
    */

    atoms_off = 0;
    for (int i = 0; i < ngpus; ++i) {
        hipSetDevice(i + 1);
        HANDLE_ERROR( hipMemcpy((void *)(atoms.data() + atoms_off), 
                                 (void *)atoms_new_dev[i], 
                                 atoms_width[i] * sizeof(float4), 
                                 hipMemcpyDeviceToHost) );
        atoms_off += atoms_width[i];
    }
    
    std::cout << "results: " << std::endl;
    for (int i = 0; i < natoms; i += natoms/10) {
        std::cout << atoms[i].x << " " << 
                     atoms[i].y << " " << 
                     atoms[i].z << " " <<
                     atoms[i].w << std::endl;
    }

    for (int i = 0; i < ngpus; ++i) {
        hipFree((void *)atoms_old_dev[i]);
        hipFree((void *)atoms_new_dev[i]);
    }

    return 0;
    
}

int main() {
    std::chrono::time_point<std::chrono::steady_clock> one_start, one_end;
    one_start = std::chrono::steady_clock::now();
    run(2, 10);
    one_end = std::chrono::steady_clock::now();
    std::chrono::duration<double> one_dur = one_end - one_start;
    
    std::chrono::time_point<std::chrono::steady_clock> two_start, two_end;
    two_start = std::chrono::steady_clock::now();
    run(1, 10);
    two_end = std::chrono::steady_clock::now();
    std::chrono::duration<double> two_dur = two_end - two_start;
    
    std::cout << "one took: " << one_dur.count() << " seconds; " << std::endl;
    std::cout << "two took: " << two_dur.count() << " seconds; " << std::endl;
}

