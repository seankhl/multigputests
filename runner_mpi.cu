#include "hip/hip_runtime.h"

#include <vector>
#include <random>
#include <algorithm>
#include <functional>
#include <iostream>
#include <chrono>
//#include "Eigen/dense"

#include "mpi.h"

#include "sim.cuh"

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

const int X_SZ = 2048;
const int Y_SZ = 2048;
const float RANGE = 32;
const int natoms = 1000;

/* Check capability of the GPU (should be done for each card to be used) */
void printDeviceCheck()
{
    int ngpus;
    hipGetDeviceCount(&ngpus);
    
    std::vector<hipDeviceProp_t> gpuprops(ngpus);
    // second argument is gpu number
    for (int i = 0; i < ngpus; ++i) {
        hipGetDeviceProperties(&gpuprops[i], i);
    }
    
    // check results
    std::cout <<
        "--------------------------------------------------------------------------------" 
    << std::endl;
    std::cout << "Devices: " << std::endl;
    
    bool is_fermi = true;
    bool has_uva = true;
    for (int i = 0; i < ngpus; ++i) {
        std::cout << "    " <<
                     gpuprops[i].name << " " <<
                     gpuprops[i].major << " " <<
                     gpuprops[i].unifiedAddressing << 
                     gpuprops[i].pciBusID << " " <<
                     gpuprops[i].pciDeviceID << std::endl;
        is_fermi &= (gpuprops[i].major >= 2);  // must be fermi or newer
        has_uva &= (gpuprops[i].unifiedAddressing);
    }
    
    // TODO: only works for ngpus == 2
    int access1from0;
    int access0from1;
    hipDeviceCanAccessPeer(&access1from0, 1, 0);
    hipDeviceCanAccessPeer(&access0from1, 0, 1);
    bool same_complex = (access1from0 && access0from1);
    
    std::cout << "Peer access: " << std::endl << 
                 "    access 1 from 0: " << access1from0 << std::endl <<
                 "    access 0 from 1: " << access0from1 << std::endl;
    
    std::cout << "General info: " << std::endl << 
                 "    num devices? " << ngpus << std::endl <<
                 "    is fermi? " << is_fermi << std::endl <<
                 "    has uva? " << has_uva << std::endl <<
                 "    same complex? " << same_complex << std::endl;
    std::cout <<
        "--------------------------------------------------------------------------------" 
    << std::endl;
}

void MPI_Sendrecv_gpulocal(
        const void *sendbuf, void *recvbuf, 
        int count, MPI_Datatype type)
{
    int mpi_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    /*
    MPI_Sendrecv(sendbuf, count, type, mpi_rank, 0,
                 recvbuf, count, type, mpi_rank, MPI_ANY_TAG,
                 MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    */
    MPI_Request requests[2];
    MPI_Status statuses[2];
    MPI_Isend(sendbuf, count, type, mpi_rank, 0,
              MPI_COMM_WORLD, &requests[0]);
    MPI_Irecv(recvbuf, count, type, mpi_rank, MPI_ANY_TAG,
              MPI_COMM_WORLD, &requests[1]);
    MPI_Waitall(2, requests, statuses);
}

int run(int ngpus_in, int nt)
{
    //constexpr int X_NBINS = X_SZ / RANGE + 1;
    //constexpr int Y_NBINS = Y_SZ / RANGE + 1;
    //Grid grid = new Grid(X_NBINS, Y_NBINS);

    int ngpus;
    hipGetDeviceCount(&ngpus);
    if (ngpus_in < ngpus) { ngpus = ngpus_in; }

    // random initial values for atoms
    std::random_device rd;
    std::mt19937 mt_rand(rd());
    std::mt19937::result_type x_seed = time(0);
    auto x_rand = std::bind(
                        std::uniform_real_distribution<float>(0, X_SZ), 
                        std::mt19937(1));
    std::mt19937::result_type y_seed = time(0);
    auto y_rand = std::bind(
                        std::uniform_real_distribution<float>(0, Y_SZ), 
                        std::mt19937(2));
    std::mt19937::result_type val_seed = time(0);
    auto val_rand = std::bind(
                        std::uniform_real_distribution<float>(0, 1), 
                        std::mt19937(3));

    // create atoms and store them in the grid
    std::array<float4, natoms> atoms;
    for (int i = 0; i < natoms; ++i) {
        atoms[i].x = x_rand();
        atoms[i].y = y_rand();
        atoms[i].z = val_rand();
        //grid.record_atom(atoms[atoms.size() - 1]);
    }

    std::sort(atoms.begin(), atoms.end(), [](float4 a, float4 b) {
        return a.y < b.y;
    });

    int atoms_off = 0;
    
    std::vector<int> cutlo(ngpus, 0);
    std::vector<int> atoms_split(ngpus, natoms);
    std::vector<int> atoms_width(ngpus, natoms);
    std::vector<int> cuthi(ngpus, natoms);
    for (int i = 0; i < ngpus; ++i) {
        for (int j = 0; j < natoms; ++j) {
            if (atoms[j].y > Y_SZ/ngpus * (i+1)) {
                atoms_split[i] = j;
                atoms_width[i] = j;
                break;
            }
        }
    }
    for (int i = 1; i < ngpus; ++i) {
        atoms_width[i] -= atoms_width[i-1];
    }
    for (int i = 0; i < ngpus; ++i) {
        for (int j = 0; j < natoms; ++j) {
            if (atoms[j].y > (Y_SZ/ngpus - RANGE) * (i+1)) {
                cutlo[i] = j;
                break;
            }
        }
    }
    for (int i = 0; i < ngpus; ++i) {
        for (int j = 0; j < natoms; ++j) {
            if (atoms[j].y > (Y_SZ/ngpus + RANGE) * (i+1)) {
                cuthi[i] = j;
                break;
            }
        }
    }
    for (int i = 0; i < ngpus; ++i) {
        std::cout << "gpu id: " << i+1 << 
                     " cutlo: " << cutlo[i] <<
                     " split: " << atoms_split[i] <<
                     " cuthi: " << cuthi[i] << std::endl;
    }

    std::vector<float4 *> atoms_old_dev(ngpus);
    std::vector<float4 *> atoms_new_dev(ngpus);
    for (int i = 0; i < ngpus; ++i) {
        hipSetDevice(i);
        
        // new vals: before timestepping, current vals always in here
        HANDLE_ERROR( hipMalloc((void **)&atoms_new_dev[i], 
                                 atoms_width[i] * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)atoms_new_dev[i], 
                                 (void *)(atoms.data() + atoms_off), 
                                 atoms_width[i] * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
        
        // malloc space for old vals
        HANDLE_ERROR( hipMalloc((void **)&atoms_old_dev[i], 
                                 atoms_width[i] * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)atoms_old_dev[i], 
                                 (void *)(atoms.data() + atoms_off), 
                                 atoms_width[i] * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
        
        atoms_off += atoms_width[i];
    }
    
    std::vector<float4 *> ghost_lo_dev(ngpus-1);
    std::vector<float4 *> ghost_hi_dev(ngpus-1);
    for (int i = 0; i < ngpus-1; ++i) {  // don't need last split, always end
        // ghost vals
        hipSetDevice(i + 1);  // next proc gets lo ghosts
        HANDLE_ERROR( hipMalloc((void **)&ghost_lo_dev[i], 
                                 (atoms_split[i] - cutlo[i]) * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)ghost_lo_dev[i], 
                                 (void *)(atoms.data() + cutlo[i]), 
                                 (atoms_split[i] - cutlo[i]) * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
        hipSetDevice(i);  // this proc gets hi ghosts
        HANDLE_ERROR( hipMalloc((void **)&ghost_hi_dev[i], 
                                 (cuthi[i] - atoms_split[i]) * sizeof(float4)) );
        HANDLE_ERROR( hipMemcpy((void *)ghost_hi_dev[i], 
                                 (void *)(atoms.data() + atoms_split[i]), 
                                 (cuthi[i] - atoms_split[i]) * sizeof(float4), 
                                 hipMemcpyHostToDevice) );
    }
    
    // timestep
    //int x_cell = 0;
    //int y_cell = 0;
    std::cout << "num timesteps: " << nt << std::endl;
    for (int i = 0; i < natoms; i += natoms/10) {
        std::cout << atoms[i].x << " " << 
                     atoms[i].y << " " << 
                     atoms[i].z << std::endl;
    }

    float4 *needs_lo;
    int needs_lo_sz;
    float4 *needs_hi;
    int needs_hi_sz;
    float4 *atoms_tmp_dev;
    for (int t = 0; t < nt; ++t) {
        for (int i = 0; i < ngpus; ++i) {
            hipSetDevice(i);
            //if (t % 10000 == 0) { std::cout << t << std::endl; }

            // figure out what our needed ghosts are
            if (i == 0) {
                needs_lo = NULL;
                needs_lo_sz = 0;
            } else {
                needs_lo = ghost_lo_dev[i-1];
                needs_lo_sz = atoms_split[i-1] - cutlo[i-1];
            }
            if (i == ngpus-1) {
                needs_hi = NULL;
                needs_hi_sz = 0;
            } else {
                needs_hi = ghost_hi_dev[i];
                needs_hi_sz = cuthi[i] - atoms_split[i];
            }
            
            // swap old and new pointers
            atoms_tmp_dev = atoms_new_dev[i];
            atoms_new_dev[i] = atoms_old_dev[i];
            atoms_old_dev[i] = atoms_tmp_dev;

            // run sim
            timestep<<<atoms_width[i], 1>>>(
                        atoms_width[i], RANGE, 
                        atoms_old_dev[i], atoms_new_dev[i],
                        needs_lo, needs_lo_sz,
                        needs_hi, needs_hi_sz);

            hipDeviceSynchronize();

            // update ghosts
            if (i != 0) {
                int ghost_lo_sz = atoms_split[i-1] - cutlo[i-1];
                if (ghost_lo_sz != 0) {  // i != ngpus-1
                    /*
                    HANDLE_ERROR( 
                        cuMemcpyAsync(
                            (void *)ghost_lo_dev[i-1], 
                            (void *)(atoms_new_dev[i-1] + 
                                        (atoms_width[i-1] - ghost_lo_sz)), 
                            ghost_lo_sz * sizeof(float4), 
                            0) );
                    */
                    /*
                    HANDLE_ERROR( 
                        hipMemcpy(
                            (void *)ghost_lo_dev[i-1], 
                            (void *)(atoms_new_dev[i-1] + 
                                        (atoms_width[i-1] - ghost_lo_sz)), 
                            ghost_lo_sz * sizeof(float4), 
                            hipMemcpyDeviceToDevice) );
                    */
                    
                    MPI_Sendrecv_gpulocal(
                            (void *)(atoms_new_dev[i-1] +
                                        (atoms_width[i-1] - ghost_lo_sz)), 
                            (void *)ghost_lo_dev[i-1],
                            ghost_lo_sz * sizeof(float4),
                            MPI_FLOAT);
                    
                }
                int ghost_hi_sz = cuthi[i-1] - atoms_split[i-1];
                if (cuthi[i-1] - atoms_split[i-1] != 0) {  // i != 0
                    HANDLE_ERROR( 
                        hipMemcpy(
                            (void *)ghost_hi_dev[i-1], 
                            (void *)(atoms_new_dev[i]), 
                            ghost_hi_sz * sizeof(float4), 
                            hipMemcpyDeviceToDevice) );
                }
            }

        }
        /*
        for (auto atom: atoms) {
            grid.get_cell_by_atom(atom, x_cell, y_cell);
            for (int x_off = -1; x_off <= 1; ++x_off) {
                for (int y_off = -1; y_off <= 1; ++y_off) {
                    vector<int> neighbors = grid.get_atoms_in_cell(
                                                x_cell + x_off, 
                                                y_cell + y_off);
                    if (x_off != 0 && y_off != 0) {
                        neighbors
        */
    }
    /*
            timestep<<<atoms_width[i], 1>>>(
                        atoms_width[i], RANGE, 
                        atoms_new_dev[i], atoms_old_dev[i],
                        ghost_lo_dev[i], atoms_split[i] - cutlo[i],
                        ghost_hi_dev[i], cuthi[i] - atoms_split[i]);
    */

    atoms_off = 0;
    for (int i = 0; i < ngpus; ++i) {
        hipSetDevice(i);
        HANDLE_ERROR( hipMemcpy((void *)(atoms.data() + atoms_off), 
                                 (void *)atoms_new_dev[i], 
                                 atoms_width[i] * sizeof(float4), 
                                 hipMemcpyDeviceToHost) );
        atoms_off += atoms_width[i];
    }
    
    std::cout << "results: " << std::endl;
    for (int i = 0; i < natoms; i += natoms/10) {
        std::cout << atoms[i].x << " " << 
                     atoms[i].y << " " << 
                     atoms[i].z << " " <<
                     atoms[i].w << std::endl;
    }

    for (int i = 0; i < ngpus; ++i) {
        hipFree((void *)atoms_old_dev[i]);
        hipFree((void *)atoms_new_dev[i]);
    }

    return 0;
    
}

int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
    printDeviceCheck();

    std::chrono::time_point<std::chrono::steady_clock> two_start, two_end;
    two_start = std::chrono::steady_clock::now();
    run(2, 10);
    two_end = std::chrono::steady_clock::now();
    std::chrono::duration<double> two_dur = two_end - two_start;
    std::cout <<
        "--------------------------------------------------------------------------------" 
    << std::endl;
    
    std::chrono::time_point<std::chrono::steady_clock> one_start, one_end;
    one_start = std::chrono::steady_clock::now();
    run(1, 10);
    one_end = std::chrono::steady_clock::now();
    std::chrono::duration<double> one_dur = one_end - one_start;
    std::cout <<
        "--------------------------------------------------------------------------------" 
    << std::endl;
    
    std::cout << "one took: " << one_dur.count() << " seconds; " << std::endl;
    std::cout << "two took: " << two_dur.count() << " seconds; " << std::endl;
    MPI_Finalize();
}

